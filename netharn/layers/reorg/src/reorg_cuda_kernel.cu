#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "reorg_cuda_kernel.h"

#define BLOCK 512

dim3 cuda_gridsize(int n)
{
    int k = (n-1) / BLOCK + 1;
    int x = k;
    int y = 1;
    if(x > 65535){
        x = ceil(sqrt(k));
        y = (n-1)/(x*BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    //printf("%ld %ld %ld %ld\n", n, x, y, x*y*BLOCK);
    return d;
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= N) return;
    int in_index = i;
    int in_w = i%w;
    i = i/w;
    int in_h = i%h;
    i = i/h;
    int in_c = i%c;
    i = i/c;
    int b = i%batch;

    int out_c = c/(stride*stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w*stride + offset % stride;
    int h2 = in_h*stride + offset / stride;
    //printf("%d\n", offset);
    int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

   // printf("%d %d %d\n", w2, h2, c2);
    //printf("%d %d\n", in_index, out_index);
    //if(out_index >= N || out_index < 0) printf("bad bad bad \n");

    if(forward) out[out_index] = x[in_index];
    else out[in_index] = x[out_index];
    //if(forward) out[1] = x[1];
    //else out[0] = x[0];
}

void reorg_ongpu(float *x, int w, int h, int c, int batch, int stride, int forward, float *out, hipStream_t stream)
{
    int size = w*h*c*batch;
    hipError_t err;

    reorg_kernel<<<cuda_gridsize(size), BLOCK, 0, stream>>>(size, x, w, h, c, batch, stride, forward, out);

    err = hipGetLastError();
    if(hipSuccess != err)
    {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
        exit( -1 );
    }
}



#ifdef __cplusplus
}
#endif
